#include "hip/hip_runtime.h"
﻿#include<iostream>
#include<fstream>
#include<sstream>
#include<string.h>
#include<Windows.h>
#include"hip/hip_runtime.h"
#include""
#include"hip/device_functions.h"
#define CUDA_CHECK(call) \
do { \
    hipError_t result = call; \
    if (result != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", \
                __FILE__, __LINE__, result, hipGetErrorString(result), #call); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

using namespace std;
#define E_LineN 756 //被消元行数 line number
#define RN 85401 //矩阵列数 row number

char* eliminer;
struct line {
    bool ifUprade;
    int num;
    char bit[RN / 8 + 1];
};
line* eline;
int flagUpgrade;
int nextstart;
void init() {

    for (int i = 0; i < E_LineN; i++) {
        for (int j = RN / 8; j >= 0; j--)
            eline[i].bit[j] = 0;
    }
    for (int i = 0; i < RN; i++) {
        for (int j = RN / 8; j >= 0; j--)
            eliminer[i * (RN / 8 + 1) + j] = 0;
    }
}
//在当前消元子不为空，传入消元子下标i，GPU开始消元工作，每个线程得到整体的索引t_id，stride是所有线程数量，作为步长
__global__ void eliminate1_kernel(char* eliminer, line* eline, int i) {
    int t_id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int j = t_id; j < E_LineN; j += stride) {
        if (eline[j].num == i) {
            int ss = 0;
            for (ss; ss <= i / 8; ss++)
                eline[j].bit[ss] ^= eliminer[i * (RN / 8 + 1) + ss];
            bool flag2 = true;
            for (int j2 = RN / 8; j2 >= 0; j2--) {
                if (!(eline[j].bit[j2] == 0)) {
                    flag2 = false;
                    break;
                }
            }
            if (flag2) {
                eline[j].ifUprade = true;
                eline[j].num = -2;
            }
            else {
                for (int n = eline[j].num; n >= 0; n--) {
                    if (eline[j].bit[n / 8] & (1 << (n % 8))) {
                        eline[j].num = n;
                        break;
                    }
                }
            }
        }
    }

}
//在当前消元子为空时，cpu首先完成升格操作，传入当前是否升格成功标志flagUpgrade
//在成功升格后，需要gpu完成剩余的被消元行的消去工作，其中nextstart是开始的被消元行的下标
__global__ void eliminate2_kernel(char* eliminer, line* eline, int i, int flagUpgrade, int nextstart) {
    int t_id = threadIdx.x + blockIdx.x * blockDim.x;//全局索引
    int stride = blockDim.x * gridDim.x;  //步长
    int newid = nextstart + 1;
    for (int j = newid + t_id; j < E_LineN; j += stride) {
        if (eline[j].num == i) {
            int ss = 0;
            for (ss; ss <= i / 8; ss++)  //第i/8个可能对不需要操作的位进行了异或
                eline[j].bit[ss] ^= eliminer[i * (RN / 8 + 1) + ss];
            bool flag2 = true;
            for (int j2 = RN / 8; j2 >= 0; j2--) {
                if (!(eline[j].bit[j2] == 0)) {
                    flag2 = false;
                    break;
                }
            }
            if (flag2) {
                eline[j].ifUprade = true;
                eline[j].num = -2;
            }
            else {//重置num
                for (int n = eline[j].num; n >= 0; n--) {//找到首个1
                    if (eline[j].bit[n / 8] & (1 << (n % 8))) {  //n%8为0-7的位置，n/8为char的位置
                        eline[j].num = n;
                        break;
                    }
                }
            }
        }
    }
}

int main() {

    hipMallocManaged(&eliminer, sizeof(char) * RN * (RN / 8 + 1));
    hipMallocManaged((void**)&eline, sizeof(line) * E_LineN);
   
    init();
    // 消元子读入
    ifstream file_eliminer;
    //file_eliminer.open("特殊高斯消去的data//测试样例1 矩阵列数130，非零消元子22，被消元行8//消元子.txt", ios_base::in);
    //file_eliminer.open("特殊高斯消去的data//测试样例2 矩阵列数254，非零消元子106，被消元行53//消元子.txt", ios_base::in);
    //file_eliminer.open("特殊高斯消去的data//测试样例3 矩阵列数562，非零消元子170，被消元行53//消元子.txt", ios_base::in);
    //file_eliminer.open("特殊高斯消去的data//测试样例4 矩阵列数1011，非零消元子539，被消元行263//消元子.txt", ios_base::in);
   // file_eliminer.open("特殊高斯消去的data//测试样例5 矩阵列数2362，非零消元子1226，被消元行453//消元子.txt", ios_base::in);
    //file_eliminer.open("特殊高斯消去的data//测试样例6 矩阵列数3799，非零消元子2759，被消元行1953//消元子.txt", ios_base::in);
    //file_eliminer.open("特殊高斯消去的data//测试样例7 矩阵列数8399，非零消元子6375，被消元行4535//消元子.txt", ios_base::in);
   // file_eliminer.open("特殊高斯消去的data//测试样例8 矩阵列数23075，非零消元子18748，被消元行14325//消元子.txt", ios_base::in);
    //file_eliminer.open("特殊高斯消去的data//测试样例9 矩阵列数37960，非零消元子29304，被消元行14291//消元子.txt", ios_base::in);
    file_eliminer.open("特殊高斯消去的data//测试样例11 矩阵列数85401，非零消元子5724，被消元行756//消元子.txt", ios_base::in);
    string s1;
    if (!file_eliminer.is_open())
        cout << " 打开失败";
    while (getline(file_eliminer, s1)) {
        stringstream st;
        st << s1;
        int tmp;
        int row;
        bool first = true;
        while (st >> tmp) {
            if (first) {
                row = tmp;
                first = false;
            }
            int i = tmp / 8;
            int j = tmp % 8;
            eliminer[row * (RN / 8 + 1) + i] |= (1 << j);
        }
    }
    file_eliminer.close();
    // 被消元行的读入
    ifstream file_eline;
    
    //file_eline.open("特殊高斯消去的data//测试样例1 矩阵列数130，非零消元子22，被消元行8//被消元行.txt", ios_base::in);
    //file_eline.open("特殊高斯消去的data//测试样例3 矩阵列数562，非零消元子170，被消元行53//被消元行.txt", ios_base::in);
    //file_eline.open("特殊高斯消去的data//测试样例4 矩阵列数1011，非零消元子539，被消元行263//被消元行.txt", ios_base::in);
   // file_eline.open("特殊高斯消去的data//测试样例5 矩阵列数2362，非零消元子1226，被消元行453//被消元行.txt", ios_base::in);
   // file_eline.open("特殊高斯消去的data//测试样例6 矩阵列数3799，非零消元子2759，被消元行1953//被消元行.txt", ios_base::in);
    //file_eline.open("特殊高斯消去的data//测试样例7 矩阵列数8399，非零消元子6375，被消元行4535//被消元行.txt", ios_base::in);
    //file_eline.open("特殊高斯消去的data//测试样例8 矩阵列数23075，非零消元子18748，被消元行14325//被消元行.txt", ios_base::in);
   // file_eline.open("特殊高斯消去的data//测试样例9 矩阵列数37960，非零消元子29304，被消元行14291//被消元行.txt", ios_base::in);
    file_eline.open("特殊高斯消去的data//测试样例11 矩阵列数85401，非零消元子5724，被消元行756//被消元行.txt", ios_base::in);
    string s2;
    if (!file_eline.is_open())
        cout << " 打开失败";
    int x = 0;
    while (getline(file_eline, s2)) {
        stringstream st;
        st << s2;
        int tmp;
        bool first = true;
        while (st >> tmp) {
            if (first) {
                eline[x].num = tmp;
                first = false;
            }
            int i = tmp / 8;
            int j = tmp % 8;
            eline[x].bit[i] |= (1 << j);
        }
        x++;
    }
    file_eline.close();

    // 创建CUDA事件
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, 0));

    for (int i = RN - 1; i >= 0; i--) {
        hipDeviceSynchronize();
        bool flag1 = (eliminer[i * (RN / 8 + 1) + i / 8] == 0) ? true : false;//判断当前消元子是否为空

        if (!flag1) { //不为空时
            eliminate1_kernel << <128, 1024 >> > (eliminer, eline, i);
        }
        else {//为空时，需要cpu升格
            flagUpgrade = 0;
            nextstart = 0;
            for (int j = 0; j < E_LineN; j++) {
                if (eline[j].num == i) {
                    for (int ss = eline[j].num / 8; ss >= 0; ss--)  //赋值过来
                        eliminer[eline[j].num * (RN / 8 + 1) + ss] = eline[j].bit[ss];
                    eline[j].ifUprade = true;
                    flagUpgrade = 1;
                    nextstart = j;//之前的被消元行的首位不等于i,不用消元
                    eline[j].num = -2;
                    break;
                }
            }
            if (flagUpgrade == 0)
                continue;
            eliminate2_kernel << <32, 512 >> > (eliminer, eline, i, flagUpgrade, nextstart);

        }
    }
    // 销毁CUDA事件
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    float elapsedTime;
    CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    //记录时间
    cout << "程序用时：" << elapsedTime << "毫秒" << endl;

    // 把最后的消元子作为结果写入消元结果中
    ofstream out;
    out.open("消元结果1.txt", ios::trunc);
    for (int i = 0; i < E_LineN; i++) {
        string s;
        string stemp;
        stringstream ss;
        for (int j = RN / 8; j >= 0; j--) {
            for (int k = 7; k >= 0; k--) {
                if (eline[i].bit[j] & (1 << k)) {
                    ss.clear();
                    ss << (j * 8 + k);
                    ss >> stemp;
                    s.append(stemp);
                    s.append(" ");
                }
            }
        }
        out << s << "\n";
        s.clear();
    }

    // 释放内存
    hipFree(eliminer);
    hipFree(eline);
    
    return 0;
}
