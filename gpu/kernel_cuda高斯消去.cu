﻿#include"hip/hip_runtime.h"

#include<stdio.h>
#include<iostream>
#include"hip/device_functions.h"
const int N = 1024;
float* m;
void reset() {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < i; j++)
			m[i * N + j] = 0;
		m[i * N + i] = 1.0;
		for (int j = i + 1; j < N; j++)
			m[i * N + j] = rand();
	}
		for (int k = 0; k < N; k++)
			for (int i = k + 1; i < N; i++)
				for (int j = 0; j < N; j++)
					m[i * N + j] += m[k * N + j];
	
}
__global__ void division_kernel(float* m, int k) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;//计算线程索引
	int element = m[k * N + k];
	int temp = m[k * N + tid];
	m[k * N + tid] = (float)temp / element;

}
//除法部分
__global__ void eliminate_kernel(float* m, int k) {
	int tk = blockDim.x * blockIdx.x + threadIdx.x;
	if (tk == 0)
		m[k * N + k] = 1.0;//对角线元素设为1
	int row = k + 1 + blockIdx.x;//每个块负责一行
	while (row < N) {
		int tid = threadIdx.x;
		while (k + 1 + tid < N) {
			int col = k + 1 + tid;
			float temp_1 = m[(row * N) + col];
			float temp_2 = m[(row * N) + k];
			float temp_3 = m[k * N + col];
			m[(row * N) + col] = temp_1 - temp_2 * temp_3;
			tid = tid + blockDim.x;
		}
		__syncthreads();//同步
		if (threadIdx.x == 0)
			m[row * N + k] = 0;
		row += gridDim.x;
	}
	return;
		
}
//消去部分
int main() {
	float timecount;
	size_t size = N * N * sizeof(float);
	hipMallocManaged(&m, size);
	reset();
	hipEvent_t start, stop;
	float elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//开始计时
	hipError_t ret;
	for (int k = 0; k < N; k++) {
		division_kernel<<<1,1024>>>(m, k);
		hipDeviceSynchronize();
		ret = hipGetLastError();
		if (ret != hipSuccess)
			printf("division_kernel failed,%s\n", hipGetErrorString(ret));
		eliminate_kernel <<<128,1024>>> (m, k);
		hipDeviceSynchronize();
		ret = hipGetLastError();
		if(ret!=hipSuccess)
			printf("division_kernel failed,%s\n", hipGetErrorString(ret));

	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipFree(m);
	printf("GPU_LU:%f ms\n", elapsedTime);
	return 0;
}